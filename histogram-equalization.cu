#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"
#include <chrono>
#include <ctime>  
#include <iostream>

void histogram(int *hist_out, unsigned char *img_in, int img_size, int nbr_bin)
{
    int i;
    for (i = 0; i < nbr_bin; i++)
    {
        hist_out[i] = 0;
    }

    for (i = 0; i < img_size; i++)
    {
        hist_out[img_in[i]]++;
    }
}

void histogram_equalization(unsigned char *img_out, unsigned char *img_in,
                            int *hist_in, int img_size, int nbr_bin)
{
    int *lut = (int *)malloc(sizeof(int) * nbr_bin);
    int i, cdf, min, d;
    /* Construct the LUT by calculating the CDF */
    cdf = 0;
    min = 0;
    i = 0;
    while (min == 0)
    {
        min = hist_in[i++];
    }
    d = img_size - min;
    for (i = 0; i < nbr_bin; i++)
    {
        cdf += hist_in[i];
        //lut[i] = (cdf - min)*(nbr_bin - 1)/d;
        lut[i] = (int)(((float)cdf - min) * 255 / d + 0.5);
        if (lut[i] < 0)
        {
            lut[i] = 0;
        }
    }

    /* Get the result image */
    for (i = 0; i < img_size; i++)
    {
        if (lut[img_in[i]] > 255)
        {
            img_out[i] = 255;
        }
        else
        {
            img_out[i] = (unsigned char)lut[img_in[i]];
        }
    }
}

__global__ void histogram_gpu(int *hist_out, unsigned char *img_in, int img_size, int nbr_bin)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __syncthreads();
    if (idx<img_size){
        if (img_in[idx]<nbr_bin){
            atomicAdd(hist_out+img_in[idx], 1);
        }
    }
    __syncthreads();
}

std::chrono::duration<double> histogram_equalization_gpu(unsigned char *img_out, unsigned char *img_in,
                            int *hist_in, int img_size, int nbr_bin)
{
    auto start = std::chrono::system_clock::now();
    int *lut = (int *)malloc(sizeof(int) * nbr_bin);
    int i, cdf, min, d;
    /* Construct the LUT by calculating the CDF */
    cdf = 0;
    min = 0;
    i = 0;
    while (min == 0)
    {
        min = hist_in[i++];
    }
    d = img_size - min;
    for (i = 0; i < nbr_bin; i++)
    {
        cdf += hist_in[i];
        //lut[i] = (cdf - min)*(nbr_bin - 1)/d;
        lut[i] = (int)(((float)cdf - min) * 255 / d + 0.5);
        if (lut[i] < 0)
        {
            lut[i] = 0;
        }
    }
    auto end = std::chrono::system_clock::now();

    std::chrono::duration<double> elapsed_seconds = end-start;

    
    int* d_lut;
    unsigned char * d_img_out;
    unsigned char * d_img_in;

    hipMalloc((void **) &d_img_out, img_size*sizeof(unsigned char));
    hipMalloc((void **) &d_lut, sizeof(int) * nbr_bin);
    hipMalloc((void **) &d_img_in, img_size*sizeof(unsigned char));

    hipMemcpy(d_img_in, img_in, img_size*sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_lut, lut, sizeof(int) * nbr_bin, hipMemcpyHostToDevice);
    
    start = std::chrono::system_clock::now();
    
    int no_blocks = img_size*sizeof(unsigned char)/1024+1;
    get_result_image<<<no_blocks,1024>>>(d_lut, d_img_out, d_img_in, img_size);
    
    end = std::chrono::system_clock::now();
    elapsed_seconds += end-start;
    
    hipMemcpy(img_out, d_img_out, img_size*sizeof(unsigned char), hipMemcpyDeviceToHost);
    return elapsed_seconds;
}


__global__ void get_result_image(int* lut, unsigned char *img_out, 
                                unsigned char *img_in, int img_size)
{
    /* Get the result image */
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (lut[img_in[i]] > 255)
    {
        img_out[i] = 255;
    }
    else
    {
        img_out[i] = (unsigned char)lut[img_in[i]];
    }

}